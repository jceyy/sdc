#include "hip/hip_runtime.h"
#include "masktemp.h"


/*****************************************************************************/
/*************************Heat source at line 201*****************************/
/*****************************************************************************/
masktemp::masktemp(std::vector<int> dimension) : tempadded(false){

    const int num_x = 2*(dimension.at(0)-1);
    const int num_y = dimension.at(1);

    // Allocate mask memory
    if(hipSuccess != hipMalloc((void**) &mask_tempadded, sizeof(int) * num_x * num_y)) {
        EXIT_ERROR("not able to allocate mask memory");
    }

    //create fft plan
    hipfftPlan2d(&c2r_plan, num_y, num_x, CUFFT_TYPE_C2R);
    //create inverse fft plan
    hipfftPlan2d(&r2c_plan, num_y, num_x, CUFFT_TYPE_R2C);
}

masktemp::~masktemp(){
    hipFree(mask_tempadded);

    if(tempadded) {
        delete theta_tempadded;
    
    }

    //free cuda fft plans
    hipfftDestroy(c2r_plan);
    hipfftDestroy(r2c_plan);
}

void masktemp::temp_circle(matrix_folder* theta, CUDA_FLOAT_REAL radius) {

    // Get number of modes
    std::vector<int> f_dim = theta->get_matrix(0)->get_matrix_dimension();
    const int num_x = 2 * (f_dim.at(0)-1),
              num_y = f_dim.at(1),
              num_xy = num_x * num_y;

    init_mask_physical_space_circle<<<create_grid_dim(num_xy), create_block_dim(num_xy)>>> (mask_tempadded, num_x, num_y, radius);

    temp_mask(theta);
}

void masktemp::temp_rectangle(matrix_folder* theta, CUDA_FLOAT_REAL width) {

    // Get number of modes
    std::vector<int> f_dim = theta->get_matrix(0)->get_matrix_dimension();
    const int num_x = 2 * (f_dim.at(0)-1),
              num_y = f_dim.at(1),
              num_xy = num_x * num_y;

    init_mask_physical_space_rectangle<<<create_grid_dim(num_xy), create_block_dim(num_xy)>>> (mask_tempadded, num_x, num_y, width);

    temp_mask(theta);
}

void masktemp::temp_mask(matrix_folder* theta) {

    // Clear old field
    if(tempadded) {
        delete theta_tempadded;
     
    }

    // Get number of modes
    std::vector<int> f_dim = theta->get_matrix(0)->get_matrix_dimension();
    const int mox = f_dim.at(0),
              moy = f_dim.at(1),
              moz = f_dim.at(2),
              moxy = mox * moy,
              num_elements_real = 2 * (mox-1) * moy;

    // prepare number of real modes
    std::vector<int> dim_real(3);
    dim_real[0] = 2 * (mox-1);
    dim_real[1] = moy;
    dim_real[2] = moz;
    theta_tempadded = new matrix_device_real(dim_real);


    // Transform theta
    for(int i = 0; i < moz; i++) {
        if(HIPFFT_SUCCESS != CUFFT_EXEC_C2R(c2r_plan, theta->get_matrix(0)->get_data() + i*moxy, theta_tempadded->get_data() + i*num_elements_real) ) {
            DBGSYNC();
            EXIT_ERROR2("c2r-fft failed", ::hipGetErrorString(hipGetLastError()));
        }
    }

    tempadded = true;
}


masktemp* masktemp::init_operator(std::vector<int> dimension){
    masktemp* op = new masktemp(dimension);
    return op;
}


void masktemp::calculate_operator(matrix_folder* theta){

    if(!tempadded) return;

    // build u_i^{(1)}(x,y,n,t) for i = 1,2,3
    std::vector<int> f_dim = theta->get_matrix(0)->get_matrix_dimension();
    const int mox = f_dim.at(0),    // modes in x direction
              moy = f_dim.at(1),    // modes in y direction
              moz = f_dim.at(2);    // modes in z direction
    const int moxy = mox * moy;     // modes in horizontal plane
    const int num_elements_real = 2 * (mox-1) * moy;    // number of elements in real space


    // prepare number of real modes
    std::vector<int> dim_real(3);
    dim_real[0] = 2 * (mox-1);
    dim_real[1] = moy;
    dim_real[2] = moz;
    matrix_device_real* theta_real = new matrix_device_real(dim_real);

    // Transform theta
    for(int i = 0; i < moz; i++) {
        if(HIPFFT_SUCCESS != CUFFT_EXEC_C2R(c2r_plan, theta->get_matrix(0)->get_data() + i*moxy, theta_real->get_data() + i*num_elements_real) ) {
            DBGSYNC();
            EXIT_ERROR2("c2r-fft failed", ::hipGetErrorString(hipGetLastError()));
        }
    }




    // Apply mask
    const dim3 grid_real = create_grid_dim(num_elements_real * moz);
    const dim3 block_real = create_block_dim(num_elements_real * moz);
//m change
    apply_mask<<<grid_real,block_real>>>(theta_real->get_data(), theta_tempadded->get_data(), mask_tempadded, 1./num_elements_real, num_elements_real, num_elements_real * moz);

    /*theta_real->scale_itself(1./num_elements_real);
    f_real->scale_itself(1./num_elements_real);
    g_real->scale_itself(1./num_elements_real);*/

    // Transform theta back
    for(int i = 0; i < moz; i++) {
        if(HIPFFT_SUCCESS != CUFFT_EXEC_R2C(r2c_plan, theta_real->get_data() + i*num_elements_real, theta->get_matrix(0)->get_data() + i*moxy) ) {
            DBGSYNC();
            EXIT_ERROR2("r2c-fft failed", ::hipGetErrorString(hipGetLastError()));
        }
    }


    delete theta_real;
   

}


__host__ static dim3 create_block_dim(int number_of_matrix_entries){
    dim3 block;
    block.x = MAX_NUMBER_THREADS_PER_BLOCK;
    return block;
}


__host__ static dim3 create_grid_dim(int num){
    dim3 grid;
    // grid.x = ceil(num / MAX_N...)
    grid.x = (num + MAX_NUMBER_THREADS_PER_BLOCK - 1) / MAX_NUMBER_THREADS_PER_BLOCK;
    return grid;
}

__device__ static int get_global_index(){
    return (threadIdx.x + (threadIdx.y + (threadIdx.z + (blockIdx.x + (blockIdx.y + (blockIdx.z)
            * gridDim.y) * gridDim.x) * blockDim.z) * blockDim.y) * blockDim.x);
}

__device__ static void get_current_matrix_indices(int& current_col, int& current_row, int& current_matrix,
                                                  int total_index, int columns, int rows, int matrices) {

    int xysize = rows * columns;

    current_col = (total_index % columns);
    current_row = ((total_index % xysize) / columns);
    current_matrix = ((total_index % (xysize * matrices)) / xysize);
}



__global__ static void apply_mask(CUDA_FLOAT_REAL* input_output, CUDA_FLOAT_REAL* mask_data, int* mask, CUDA_FLOAT_REAL factor, int num_xy, int num_entries){
    int total_index = get_global_index();

// check if thread is valid
    if(total_index < num_entries) {

        CUDA_FLOAT_REAL entry;
        if(mask[total_index % num_xy] == 1) {
            entry = input_output[total_index];
        } else {
            entry = (input_output[total_index]+1.5/100); //This is where the heat source will add temperature
        }
        input_output[total_index] = factor * entry;
	}
}

//q : are both n e c e s s a r y ?

// width in fraction of edge length
//not necessary for the mom
__global__ static void init_mask_physical_space_rectangle(int* mask, int columns, int rows, CUDA_FLOAT_REAL width){
    int total_index = get_global_index();
    int col_index = 0, row_index = 0, matrix_index = 0;
    get_current_matrix_indices(col_index, row_index, matrix_index, total_index, columns, rows, 1);

    // use a mask
	if(total_index < columns*rows){
		//init all with zeros
        int mask_val = 0;

        // rectangular mask
        if((col_index < width * columns)
                || (col_index > (1-width) * columns)
                || (row_index < width * rows)
                || (row_index > (1-width) * rows)) {
            mask_val = 1;
        }

        mask[total_index] = mask_val;
	}
}

// radius in fraction of max radius
__global__ static void init_mask_physical_space_circle(int* mask, int columns, int rows, CUDA_FLOAT_REAL radius){
    int total_index = get_global_index();
    int col_index = 0, row_index = 0, matrix_index = 0;
    get_current_matrix_indices(col_index, row_index, matrix_index, total_index, columns, rows, 1);

	//use a mask
	if(total_index < columns*rows){
        // init all with zeros
        int mask_val = 0;

        // circular mask
        //CUDA_FLOAT_REAL distance_from_center_sq = (col_index - 0.3 *  columns)*(col_index - 0.3 * columns) + (row_index - 0.3 * columns)*(row_index - 0.3 * rows);	//maybe change
	//m change
        CUDA_FLOAT_REAL distance_from_center_sq = (col_index - 0.2 *  columns)*(col_index - 0.2 * columns) + (row_index - 0.3 * rows)*(row_index - 0.3 * rows);	//maybe change
        int min_len = (rows < columns)?(rows):(columns);
        CUDA_FLOAT_REAL max_distance_from_center = radius * 0.5 * min_len;
        if(distance_from_center_sq > max_distance_from_center * max_distance_from_center) {
            mask_val = 1;
        }

        mask[total_index] = mask_val;
	}
}



